#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <cmath>

#define M 2048
#define W 15
#define w 3
#define threshold 80

using namespace std;

 __global__  void smoothening_kernel(float* d_filter,float* d_raw_image,float* d_hx,float* d_hy,float* d_gx,float* d_gy,float* d_smooth_image,float* d_edged_image,int block_size){                  
               int Bx = blockIdx.x;
               int By = blockIdx.y;
               int Tx = threadIdx.x;
               int Ty = threadIdx.y;
     /* defining row and column index tp parse through filters and image*/
   int rowd = By* block_size + Ty;
   int columd = Bx* block_size + Tx;
     
     /*boundaries checking*/
   int rr = rowd - W/2;
   int cc = columd - W/2;
   float acc = 0.0;
    /*convolution for smmothening*/   
      for(int k = 0; k < W; k++ ){
       for(int l = 0; l < W; l++){
          if((rr + k) >= 0 && (rr + k) < M && (cc + l) >= 0 && (cc + l) < M){
           acc += d_raw_image[(rr + k) * M + (cc + l)] * d_filter[k * W + l];
        }  
      }
   d_smooth_image[rowd * M + columd] = acc;
    }

    /*convolution for edge detection */
     int mm = rowd - w/2;
     int nn = columd - w/2;
     float acc1 = 0.0;
     float acc2 = 0.0;
   for(int k = 0; k < w; k++ ){
    for(int l = 0; l < w; l++){
          if((mm + k) >= 0 && (mm + k) < M && (nn + l) >= 0 && (nn + l) < M){
           acc1 += d_smooth_image[(mm + k) * M + (nn + l)] * d_hx[k * w + l];
           acc2 += d_smooth_image[(mm + k) * M + (nn + l)] * d_hy[k * w + l];
         }
      } 
   d_gx[rowd * M + columd] = acc1;
   d_gy[rowd * M + columd] = acc2;
    }
 
  // gradient magnitude of spatial domains
    d_edged_image[rowd * M + columd] = sqrt(pow(d_gx[rowd * M + columd], 2) + pow(d_gy[rowd * M + columd], 2));
    if(d_edged_image[rowd * M + columd] > threshold){d_edged_image[rowd * M + columd]  = 255;}
      else{d_edged_image[rowd * M + columd] = 0;}
     }


 int main(int argc, char* argv[]){

int block_size = atoi(argv[1]);

float h_filter[W][W];  //Gaussian filter
float h_x[w][w] = {{-1.0,0.0,1.0},{-2.0,0.0,2.0},{-1.0,0.0,1.0}}; // Sobel operator
float h_y[w][w] = {{-1.0,-2.0,-1.0},{0.0,0.0,0.0},{1.0,2.0,1.0}}; //Sobel operator
 
 double sigma = 1.5;       
 float P = 1.0/(2* M_PI * sigma*sigma);
 float Q = 2.0* M_PI * sigma*sigma;
 float sum = 0.0;
 long image_size;
 size_t elements;
 int L = (W-1)/2;

  /*initializing gaussian filter*/
 for(int x = -W/2; x <= W/2; x++){
  for(int y = -W/2; y <= W/2; y++){
    int I = (x+ W/2) - L;
    int J = (y+ W/2) - L;
    h_filter[x + W/2][y + W/2] = P*(exp(-(I*I + J*J)/Q));
    sum += h_filter[x + W/2][y + W/2];
   }
  }

 for(int i = 0; i < W; i++){
  for(int j = 0; j < W; j++){
  h_filter[i][j]/= sum;
  }
 }
  // verify gaussian filter
  cout<<"guassian filter" <<endl;
  for(int q = 0; q < 15; q++){
   for(int z = 0; z <15; z++){
     cout<<h_filter[q][z]<<" ";
    }
 cout<<endl;
  }
 cout<<" "<<endl;

FILE* fp_in, *fp_out1, *fp_out2;
   fp_in = fopen ("Rainier2048_noise.bin","rb");
      if(fp_in == NULL){cout<<"FILE ERROR!"<<endl;
          exit(1); }

   //obtain file size
  fseek(fp_in, 0, SEEK_END);
  image_size = ftell(fp_in);
  rewind(fp_in);

  // allocate buffer of image size
  unsigned char* buffer = (unsigned char*)malloc(sizeof(unsigned char) * image_size);
  unsigned char* buffer1 = (unsigned char*)malloc(sizeof(unsigned char) * image_size);
 //copy file into buffer
 elements = fread(buffer, sizeof(unsigned char), image_size, fp_in);
 if(elements != image_size){cout<<"READ ERROR! "<<endl;
       exit(2);}

  fclose(fp_in);

  float* fptr = (float*)malloc(sizeof(float)* M * M);
  
  //typecast from char to float
   for(int row = 0; row < M; row++){
   for(int col = 0; col < M; col++){
    fptr[row * M + col] = (float) buffer[row * M + col];
  }
 }

   cout<<"raw image" <<endl;
  for(int q = 1024; q < 1034; q++){
   for(int z = 1525; z <1535; z++){
     cout<<buffer[q * M + z]<<" ";
    }
 cout<<endl;
  }

  cout<<"raw image of float type" <<endl;
  for(int q = 1024; q < 1034; q++){
   for(int z = 1525; z <1535; z++){
     cout<<fptr[q * M + z]<<" ";
    }
 cout<<endl;
  }
cout<<" "<<endl;
 
 float* smooth_image = (float*)malloc(sizeof(float)* M * M);
 float* edged_image = (float*)malloc(sizeof(float)* M * M); 

   
  float* d_gx;
  float* d_gy;
  float* d_hx;
  float* d_hy;
  float* d_raw_image;
  float* d_filter;
  float* d_smooth_image;
  float* d_edged_image;

  hipMalloc((void**)&d_hx,sizeof(float)* w * w);
  hipMalloc((void**)&d_hy,sizeof(float)* w * w);
  hipMalloc((void**)&d_filter,sizeof(float)* W * W);
  hipMalloc((void**)&d_raw_image,sizeof(float)* M * M);
  hipMalloc((void**)&d_smooth_image,sizeof(float)* M * M);
  hipMalloc((void**)&d_edged_image,sizeof(float)* M * M);
  hipMalloc((void**)&d_gx,sizeof(float)* M * M);
  hipMalloc((void**)&d_gy,sizeof(float)* M * M);
 
 /* measuring execution time */
 hipEvent_t start, stop;
 hipEventCreate(&start);
 hipEventCreate(&stop);

  /*copy image and filters from host to device */
   hipMemcpy(d_raw_image, fptr, sizeof(float) * M * M, hipMemcpyHostToDevice);
   hipMemcpy(d_filter,h_filter , sizeof(float) * W * W, hipMemcpyHostToDevice);
   hipMemcpy(d_hx, h_x , sizeof(float) * w * w, hipMemcpyHostToDevice);
   hipMemcpy(d_hy, h_y , sizeof(float) * w * w, hipMemcpyHostToDevice);

  /*define block size and grid size and invoke kernel*/
    dim3 threadsPerBlock(block_size, block_size);
    int numblocks = M / block_size;
    dim3 blocksPerGrid(numblocks, numblocks);

   hipEventRecord(start);
 smoothening_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_filter,d_raw_image,d_hx,d_hy,d_gx,d_gy,d_smooth_image,d_edged_image,block_size);

     /* copy results from device to host */
  hipMemcpy(smooth_image, d_smooth_image, sizeof(float) * M * M, hipMemcpyDeviceToHost);
  hipMemcpy(edged_image, d_edged_image, sizeof(float) * M * M, hipMemcpyDeviceToHost);

 hipEventRecord(stop);
 hipEventSynchronize(stop);

float milliseconds = 0.0;

 hipEventElapsedTime(&milliseconds, start, stop);

 cout<<"he parallel execution time for block size "<< block_size << " is "<< milliseconds <<" secs" << endl;

  /* write edge detected image to file*/
   for(int row = 0; row < M; row++){
   for(int col = 0; col < M; col++){
    buffer[row * M + col] = (unsigned char) smooth_image[row * M + col];
    buffer1[row * M + col] = (unsigned char) edged_image[row * M + col];
   }
  }
     cout<<"smoothened_image buffered"<<endl;
    for(int ir = 1024; ir < 1034; ir++){
      for(int ic = 1525; ic < 1535; ic++){
    cout<< *(buffer + ir * M + ic) <<" ";
   }
 cout<<endl;
 }
 cout<<" "<<endl;

 
  
   fp_out1 = fopen("smoothened_image_cuda.bin", "wb");
   fwrite(buffer, sizeof(unsigned char), image_size, fp_out1);
    fclose(fp_out1);

   fp_out2 = fopen("Edge_detected_image_cuda.bin", "wb");
   fwrite(buffer1,sizeof(unsigned char), image_size, fp_out2);
    fclose(fp_out2);
 
   cout<<"smoothened image" <<endl;
  for(int q = 1024; q < 1034; q++){
   for(int z = 1525; z <1535; z++){
     cout<<smooth_image[q * M + z]<<" ";
    }
 cout<<endl;
  }
cout<<" "<<endl;

   cout<<"edged_image buffered"<<endl;
  for(int ir = 1024; ir < 1034; ir++){
    for(int ic = 1525; ic < 1535; ic++){
   cout<< *(buffer1 + ir * M + ic) <<" ";
   }
 cout<<endl;
 }
 cout<<" "<<endl;

   cout<<"edged_image" <<endl;
  for(int q = 1024; q < 1034; q++){
   for(int z = 1525; z <1535; z++){
     cout<<edged_image[q * M + z]<<" ";
    }
 cout<<endl;
  }
cout<<" "<<endl;

/* free device memory*/
hipFree(d_raw_image);
hipFree(d_hx);
hipFree(d_hy);
hipFree(d_smooth_image);
hipFree(d_edged_image);
hipFree(d_gx);
hipFree(d_gy);
hipFree(d_filter);

/*free host memory*/
delete[] fptr;
delete[] smooth_image;
delete[] buffer;
delete[] buffer1;
delete[] edged_image;


 return 0;
 }
