#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <ctime>
#include <sys/time.h>

using namespace std;

     /*the kernel code to run on the GPU device */
    __global__ 
          void matrix_mult_kernel(float* A, float* B, float* C, int M, int block_size){
           /* using specified conventions*/
               int Bx = blockIdx.x;
               int By = blockIdx.y;
               int Tx = threadIdx.x;
               int Ty = threadIdx.y;
               int Grid_size = M / block_size;
             
             
      /* defining row and column index tp parse through matrix A & B */
   int rowd = (By * block_size) + Ty;
   int columd = (Bx * block_size) + Tx;
   int rowds;
   int columds;
   extern  __shared__ float smem[];
                       float* ads = (float*)(smem);
                       float*  bds =(float*)(smem + block_size * block_size) ;
                       
  float tempsum = 0;

        ads[Ty * block_size + Tx] = 0;
        bds[Ty * block_size + Tx] = 0;
        

  

        /* copy data into the allocated shared memory */
        for(int k = 0; k < Grid_size; k++){
         rowds = rowd * M + (k * block_size + Tx);
         columds = (k * block_size + Ty) * M + columd;
        ads[Ty * block_size + Tx] =  A[rowds];
        bds[Ty * block_size + Tx] = B[columds];   
    
     __syncthreads();   // synchronize 

     for(int i = 0; i < block_size; i++)
     tempsum += ads[Ty * block_size + i] * bds[i * block_size + Tx];    
     
   __syncthreads();         
     }
   C[rowd * M + columd] = tempsum;
   
  } 
  


 int main(int argc, char* argv[]){ 
  int M = 4096;
  int B = atoi(argv[1]);   //block size
      
 
 /*allocate matrixes A, B, C in host memory*/ 
  float* ahptr = (float*)malloc(sizeof(float)* M * M);
  float* bhptr = (float*)malloc(sizeof(float)* M * M);
  float* chptr = (float*)malloc(sizeof(float)* M * M);
  float* dhptr = (float*)malloc(sizeof(float)* M * M);
 /* initialize matrices a, b in host memory*/
  
  for(int i = 0; i < M; i++){
   for(int j = 0; j < M; j++){
     *(ahptr + i * M + j) = ((i+1)*(j+1))/(float)M;
     *(bhptr + i * M + j) = (float)(j+1)/(i+1); 
     *(chptr +i * M + j) = 0;
     *(dhptr +i * M + j) = (i+1)*(j+1);
   }
 }
   
     //verify result   
   cout<<"result verifier"<<endl;
    for(int w = 2044; w < 2052; w++){
     for(int s = 0; s < 8; s++){
     cout<< *(dhptr + w * M + s)<<" ";
     }
   cout<<endl;
   }
  cout<<" "<<endl;

    /*allocate memoryon the device*/
 
   float* ad;
   float* bd;
   float* cd;

  hipMalloc((void**)&ad,sizeof(float)* M * M);
  hipMalloc((void**)&bd,sizeof(float)* M * M); 
  hipMalloc((void**)&cd,sizeof(float)* M* M);
   /*set shared memory to 48kB and Li cache to 16kB*/
  hipDeviceSetCacheConfig(hipFuncCachePreferShared);  
   /* measuring the execution time*/
 hipEvent_t start, stop;
 hipEventCreate(&start);
 hipEventCreate(&stop);
    /*copy matrices from host to device */
   hipMemcpy(ad, ahptr, sizeof(float) * M * M, hipMemcpyHostToDevice);
   hipMemcpy(bd, bhptr, sizeof(float) * M * M, hipMemcpyHostToDevice);

     /*invoking the kernel */
     int block_size = B;
    dim3 threadsPerBlock(block_size, block_size);
    int numblocks = M / block_size;
    dim3 blocksPerGrid(numblocks, numblocks);

 hipEventRecord(start);
 matrix_mult_kernel<<< blocksPerGrid, threadsPerBlock, sizeof(float) * block_size * block_size * 2 >>>(ad, bd, cd, M, B);

   /* copy result from device to host */
 hipMemcpy(chptr, cd, sizeof(float) * M * M, hipMemcpyDeviceToHost);

 hipEventRecord(stop);
 hipEventSynchronize(stop);

float milliseconds = 0.0;
 
 hipEventElapsedTime(&milliseconds, start, stop);

 cout<<"the parallel execution time for block size "<< B << " is "<< milliseconds <<" msecs" <<endl; 

    /*print a section of the result to verify result*/ 
   cout<<" a section of the GPU result;"<<endl;
    for(int h = 2044; h < 2052; h++){
   for(int t = 0; t < 8; t++){
   cout<< *(chptr + h * M + t) <<" ";
  }
  cout<<endl;
 }
 cout<<" "<<endl;

 /* free device memory*/
hipFree(ad);
hipFree(bd);
hipFree(cd);

 /*free host memory*/
delete [] ahptr;
delete [] bhptr;
delete [] chptr;

 return 0;
 
 };
