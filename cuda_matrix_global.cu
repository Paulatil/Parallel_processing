#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <ctime>
#include <sys/time.h>

using namespace std;

     /*the kernel code to run on the GPU device */
    __global__ 
          void matrix_mult_kernel(float* A, float* B, float* C, int M, int block_size){
           /* using specified conventions*/
               int Bx = blockIdx.x;
               int By = blockIdx.y;
               int Tx = threadIdx.x;
               int Ty = threadIdx.y;
     /* defining row and column index tp parse through matrix A & B */
   int rowd = (By * block_size) + Ty;
   int columd = (Bx * block_size) + Tx;
    float tempsum = 0;
    if(rowd < M && columd < M){
       for(int i = 0; i < M; i++){
        tempsum += A[rowd * M + i] * B[i * M + columd];
      }
    C[rowd * M + columd] = tempsum;
   } 
  }

 int main(int argc, char* argv[]){ 
  int M = 4096;
  int B = atoi(argv[1]);   //block size
      
 
 /*allocate matrixes A, B, C in host memory*/ 
  float* ahptr = (float*)malloc(sizeof(float)* M * M);
  float* bhptr = (float*)malloc(sizeof(float)* M * M);
  float* chptr = (float*)malloc(sizeof(float)* M * M);
  float* dhptr = (float*)malloc(sizeof(float)* M * M);
 /* initialize matrices a, b in host memory*/
  
  for(int i = 0; i < M; i++){
   for(int j = 0; j < M; j++){
     *(ahptr + i * M + j) = ((i+1)*(j+1))/(float)M;
     *(bhptr + i * M + j) = (float)(j+1)/(i+1); 
     *(chptr +i * M + j) = 0;
     *(dhptr +i * M + j) = (i+1)*(j+1);
   }
 }
   
     //verify result   
   cout<<"result verifier"<<endl;
    for(int w = 2044; w < 2052; w++){
     for(int s = 0; s < 8; s++){
     cout<< *(dhptr + w * M + s)<<" ";
     }
   cout<<endl;
   }
  cout<<" "<<endl;

    /*allocate memoryon the device*/
 
   float* ad;
   float* bd;
   float* cd;

  hipMalloc((void**)&ad,sizeof(float)* M * M);
  hipMalloc((void**)&bd,sizeof(float)* M * M); 
  hipMalloc((void**)&cd,sizeof(float)* M* M);

   /* measuring the execution time*/
 hipEvent_t start, stop;
 hipEventCreate(&start);
 hipEventCreate(&stop);
    /*copy matrices from host to device */
   hipMemcpy(ad, ahptr, sizeof(float) * M * M, hipMemcpyHostToDevice);
   hipMemcpy(bd, bhptr, sizeof(float) * M * M, hipMemcpyHostToDevice);

     /*invoking the kernel */
     int block_size = B;
    dim3 threadsPerBlock(block_size, block_size);
    int numblocks = M / block_size;
    dim3 blocksPerGrid(numblocks, numblocks);

 hipEventRecord(start);
 matrix_mult_kernel<<<blocksPerGrid, threadsPerBlock>>>(ad, bd, cd, M, B);

   /* copy result from device to host */
 hipMemcpy(chptr, cd, sizeof(float) * M * M, hipMemcpyDeviceToHost);

 hipDeviceSynchronize();
 hipEventRecord(stop);
 hipEventSynchronize(stop);

float milliseconds = 0.0;
 
 hipEventElapsedTime(&milliseconds, start, stop);

 cout<<"the parallel execution time for block size "<< B << " is "<< milliseconds << endl; 

    /*print a section of the result to verify result*/ 
   cout<<" a section of the GPU result;"<<endl;
    for(int h = 2044; h < 2052; h++){
   for(int t = 0; t < 8; t++){
   cout<< *(chptr + h * M + t) <<" ";
  }
  cout<<endl;
 }
 cout<<" "<<endl;

 /* free device memory*/
hipFree(ad);
hipFree(bd);
hipFree(cd);

 /*free host memory*/
delete [] ahptr;
delete [] bhptr;
delete [] chptr;

 return 0;
 
 };
